#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <fstream>
#include <cmath>
#include <mpi.h>

#define INFINITE 1000000000
int* distance_host;
int* distance_dev;


__global__ void FW1(int *distance_dev, size_t r, int vertexPadded){

    size_t i = r * blockDim.x + threadIdx.y;
    size_t j = r * blockDim.x + threadIdx.x;
    size_t offset = i * vertexPadded;

    extern __shared__ int dist[];
    dist[threadIdx.y * blockDim.x + threadIdx.x] = distance_dev[ offset + j ];
    __syncthreads();

    for(size_t k=0; k<blockDim.x; ++k){
        dist[ threadIdx.y * blockDim.x + threadIdx.x ] = min(dist[ threadIdx.y * blockDim.x + threadIdx.x ], dist[ threadIdx.y * blockDim.x + k ] + dist[ k * blockDim.x + threadIdx.x ]);
        __syncthreads();
    }
    distance_dev[ offset + j ] = dist[threadIdx.y * blockDim.x + threadIdx.x];
}


__global__ void FW2(int *distance_dev, size_t r, int vertexPadded, size_t total_round){

    size_t block_i, block_j;
    if(blockIdx.y == 0){
        block_i = r;
        block_j = (blockIdx.x + r + 1) % total_round;
    }else{
        block_j = r;
        block_i = (blockIdx.x + r + 1) % total_round;
    }

    size_t i = block_i * blockDim.x + threadIdx.y;
    size_t j = block_j * blockDim.x + threadIdx.x;
    size_t offset = i * vertexPadded;
    size_t index = threadIdx.y * blockDim.x + threadIdx.x;
    size_t blockSize_squard = blockDim.x * blockDim.x;


    extern __shared__ int dist[];
    dist[index] = distance_dev[offset + j];
    dist[blockSize_squard + index] = distance_dev[offset + threadIdx.x + r * blockDim.x];
    dist[2*blockSize_squard + index] = distance_dev[(threadIdx.y + r * blockDim.x) * vertexPadded + j ];
    __syncthreads();

    
    for (size_t k = 0; k < blockDim.x; k++) {
        size_t ik = threadIdx.y * blockDim.x + blockSize_squard + k;
        size_t kj = k * blockDim.x + 2 * blockSize_squard + threadIdx.x;
        dist[index] = min(dist[index], dist[ik] + dist[kj]);
        __syncthreads();
    }
    distance_dev[offset + j] = dist[index];
}


__global__ void FW3(int *distance_dev, size_t r, int vertexPadded, size_t total_round, int BlockOffset, int id){

    size_t block_i = blockIdx.y + BlockOffset;
    size_t block_j = (r + blockIdx.x + 1) % total_round;
    size_t i = block_i * blockDim.x + threadIdx.y;
    size_t j = block_j * blockDim.x + threadIdx.x;
    size_t offset = i * vertexPadded;

    size_t index = threadIdx.y * blockDim.x + threadIdx.x;
    size_t blockSize_squard = blockDim.x * blockDim.x;

    extern __shared__ int dist[];
    dist[index] = distance_dev[offset + j]; //block(i,j)
    dist[blockSize_squard + index] = distance_dev[offset + threadIdx.x + r * blockDim.x]; //block(i,r)
    dist[2*blockSize_squard + index] = distance_dev[(threadIdx.y + r * blockDim.x) * vertexPadded + j ]; //block(r,j)
    __syncthreads();

    
    for(size_t k = 0; k < blockDim.x; k++) {
        size_t ik = threadIdx.y * blockDim.x + blockSize_squard + k;
        size_t kj = k * blockDim.x + 2 * blockSize_squard + threadIdx.x;
        dist[index] = min(dist[index], dist[ik] + dist[kj]);
        __syncthreads();
    }
    distance_dev[offset + j] = dist[index];
}



int main(int argc, char **argv){

	int size, rank;
    MPI_Init(&argc, &argv);
    MPI_Comm_size(MPI_COMM_WORLD, &size);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    //get number of threads per block
    // int deviceNum;
    hipSetDevice(rank);
    // hipGetDeviceCount(&deviceNum);
    // hipDeviceProp_t prop;
    // hipGetDeviceProperties(&prop, 0);
    // int ThreadsPerBlock = (int) sqrt(prop.maxThreadsPerBlock);
    // int blockSize = ThreadsPerBlock;
    int deviceNum = 2;
    int blockSize = 32;
    
    
    //read input file
    std::ifstream inputFile(argv[1], std::ios::in | std::ios::binary);
    unsigned vertexNum, edgeNum;
    inputFile.read((char*)&vertexNum, 4);
    inputFile.read((char*)&edgeNum, 4);

    //calculate block number, vertex number in a block
    if(vertexNum < blockSize) blockSize = vertexNum;
    int blockNum = ceil( 1.0 * vertexNum / blockSize);
    int vertexPadded = blockSize * blockNum;


    //Allocate memory (pinned)
    hipHostMalloc(&distance_host, sizeof(int) * vertexPadded * vertexPadded);

    for(unsigned i=0; i<vertexPadded; ++i){
        for(unsigned j=0; j<vertexPadded; ++j){
			if( i!=j || i>=vertexNum || j>=vertexNum) distance_host[ i * vertexPadded + j ] = INFINITE;
            else distance_host[ i * vertexPadded + j ] = 0;
        }
    }

    int source, destination, weight;
    while( inputFile.read((char*)&source, 4) ){
        inputFile.read((char*)&destination, 4);
        inputFile.read((char*)&weight, 4);
        distance_host[ source * vertexPadded + destination ] = weight;
    }
    inputFile.close();

    
    //Blocked APSP
	size_t task_BlockNum, task_rBegin;
	size_t round = vertexPadded / blockSize;
	if(!rank){
		task_rBegin = round / deviceNum;
		task_BlockNum = task_rBegin + round % deviceNum;
	}else{
		task_rBegin = 0;
		task_BlockNum = round/deviceNum;
	}
	size_t task_rEnd = task_rBegin + task_BlockNum;
	size_t offset = task_rBegin * blockSize * vertexPadded;

    int blockRowSize = vertexPadded * blockSize;
	dim3 block(blockSize, blockSize);
    dim3 grid2(round-1, 2);
	dim3 grid3(round-1, task_BlockNum);
	
	hipMalloc(&distance_dev, sizeof(int) * vertexPadded * vertexPadded);
    hipMemcpy(&distance_dev[offset], &distance_host[offset], sizeof(int) * blockRowSize * task_BlockNum, hipMemcpyHostToDevice);

	for (size_t r = 0; r < round; ++r) {

		size_t index = r * blockSize * vertexPadded;
		if(r >= task_rBegin && r < task_rEnd){
			hipMemcpy(&distance_host[index], &distance_dev[index], sizeof(int) * blockRowSize, hipMemcpyDeviceToHost);
			MPI_Send(&distance_host[index], blockRowSize, MPI_INT, !rank, 0, MPI_COMM_WORLD);
		}else{
			MPI_Recv(&distance_host[index], blockRowSize, MPI_INT, !rank, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
			hipMemcpy(&distance_dev[index], &distance_host[index], sizeof(int) * blockRowSize, hipMemcpyHostToDevice);
		}

		FW1<<< 1, block, blockSize * blockSize * sizeof(int) >>>(distance_dev, r, vertexPadded);
		FW2<<< grid2, block, 3 * blockSize * blockSize * sizeof(int) >>>(distance_dev, r, vertexPadded, round);
		FW3<<< grid3, block, 3 * blockSize * blockSize * sizeof(int) >>>(distance_dev, r, vertexPadded, round, task_rBegin, rank);
		// hipDeviceSynchronize();
	}
	hipMemcpy( &distance_host[offset], &distance_dev[offset], sizeof(int) * blockRowSize * task_BlockNum, hipMemcpyDeviceToHost);


	if(rank) MPI_Send(&distance_host[0], blockRowSize * task_BlockNum, MPI_INT, 0, 0, MPI_COMM_WORLD);
	else{
		MPI_Recv(&distance_host[0], blockRowSize * round/deviceNum, MPI_INT, 1, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
		//write answer to output file
		std::ofstream outputFile(argv[2], std::ios::out | std::ios::binary);
		for(int i=0; i<vertexNum; ++i){
			for(int j=0; j<vertexNum; ++j){
				outputFile.write( (char*)&distance_host[ i * vertexPadded + j ], 4);
			}
		}
		outputFile.close();
	}
    
    hipFree(distance_host);
	hipFree(distance_dev);
	
	MPI_Finalize(); 
    return 0;
}