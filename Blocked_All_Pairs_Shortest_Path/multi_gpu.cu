
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <cmath>
#include <omp.h>

#define INFINITE 1000000000
int* distance_host;
int* distance_dev[2];


__global__ void FW1(int *distance_dev, int r, int vertexPadded){

    int i = r * blockDim.x + threadIdx.y;
    int j = r * blockDim.x + threadIdx.x;
    int offset = i * vertexPadded;

    extern __shared__ int dist[];
    dist[threadIdx.y * blockDim.x + threadIdx.x] = distance_dev[ offset + j ];
    __syncthreads();

    for(int k=0; k<blockDim.x; ++k){
        dist[ threadIdx.y * blockDim.x + threadIdx.x ] = min(dist[ threadIdx.y * blockDim.x + threadIdx.x ], dist[ threadIdx.y * blockDim.x + k ] + dist[ k * blockDim.x + threadIdx.x ]);
        __syncthreads();
    }
    distance_dev[ offset + j ] = dist[threadIdx.y * blockDim.x + threadIdx.x];
}


__global__ void FW2(int *distance_dev, int r, int vertexPadded, int total_round){

    int block_i, block_j;
    if(blockIdx.y == 0){
        block_i = r;
        block_j = (blockIdx.x + r + 1) % total_round;
    }else{
        block_j = r;
        block_i = (blockIdx.x + r + 1) % total_round;
    }

    int i = block_i * blockDim.x + threadIdx.y;
    int j = block_j * blockDim.x + threadIdx.x;
    int offset = i * vertexPadded;
    int index = threadIdx.y * blockDim.x + threadIdx.x;
    int blockSize_squard = blockDim.x * blockDim.x;


    extern __shared__ int dist[];
    dist[index] = distance_dev[offset + j];
    dist[blockSize_squard + index] = distance_dev[offset + threadIdx.x + r * blockDim.x];
    dist[2*blockSize_squard + index] = distance_dev[(threadIdx.y + r * blockDim.x) * vertexPadded + j ];
    __syncthreads();

    
    for (int k = 0; k < blockDim.x; k++) {
        int ik = threadIdx.y * blockDim.x + blockSize_squard + k;
        int kj = k * blockDim.x + 2 * blockSize_squard + threadIdx.x;
        dist[index] = min(dist[index], dist[ik] + dist[kj]);
        __syncthreads();
    }
    distance_dev[offset + j] = dist[index];
}


__global__ void FW3(int *distance_dev, int r, int vertexPadded, int total_round, int BlockOffset, int id){

    int block_i = blockIdx.y + BlockOffset;
    int block_j = blockIdx.x;
    int i = block_i * blockDim.x + threadIdx.y;
    int j = block_j * blockDim.x + threadIdx.x;
    int offset = i * vertexPadded;

    int index = threadIdx.y * blockDim.x + threadIdx.x;
    int blockSize_squard = blockDim.x * blockDim.x;

    extern __shared__ int dist[];
    dist[index] = distance_dev[offset + j]; //block(i,j)
    dist[blockSize_squard + index] = distance_dev[offset + threadIdx.x + r * blockDim.x]; //block(i,r)
    dist[2*blockSize_squard + index] = distance_dev[(threadIdx.y + r * blockDim.x) * vertexPadded + j ]; //block(r,j)
    __syncthreads();

    
    for (int k = 0; k < blockDim.x; k++) {
        int ik = threadIdx.y * blockDim.x + blockSize_squard + k;
        int kj = k * blockDim.x + 2 * blockSize_squard + threadIdx.x;
        dist[index] = min(dist[index], dist[ik] + dist[kj]);
        __syncthreads();
    }
    distance_dev[offset + j] = dist[index];
}



int main(int argc, char **argv){

    //get number of threads per block
    int deviceNum;
    hipSetDevice(0);
    hipGetDeviceCount(&deviceNum);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int ThreadsPerBlock = (int) sqrt(prop.maxThreadsPerBlock);
    int blockSize = ThreadsPerBlock;
    
    
    //read input file
    std::ifstream inputFile(argv[1], std::ios::in | std::ios::binary);
    unsigned vertexNum, edgeNum;
    inputFile.read((char*)&vertexNum, 4);
    inputFile.read((char*)&edgeNum, 4);

    //calculate block number, vertex number in a block
    if(vertexNum < blockSize) blockSize = vertexNum;
    int blockNum = ceil( 1.0 * vertexNum / blockSize);
    int vertexPadded = blockSize * blockNum;


    //Allocate memory (pinned)
    hipHostMalloc(&distance_host, sizeof(int) * vertexPadded * vertexPadded, hipHostMallocDefault);

    for(unsigned i=0; i<vertexPadded; ++i){
        for(unsigned j=0; j<vertexPadded; ++j){
            if( i>=vertexNum || j>=vertexNum) distance_host[ i * vertexPadded + j ] = INFINITE;
            else if( i == j) distance_host[ i * vertexPadded + j ] = 0;
            else distance_host[ i * vertexPadded + j ] = INFINITE;
        }
    }

    int source, destination, weight;
    while( inputFile.read((char*)&source, 4) ){
        inputFile.read((char*)&destination, 4);
        inputFile.read((char*)&weight, 4);
        distance_host[ source * vertexPadded + destination ] = weight;
    }
    inputFile.close();

    
    int round = vertexPadded / blockSize;
    dim3 block(blockSize, blockSize);
    dim3 grid2(round-1, 2);

    #pragma omp parallel num_threads(deviceNum)
    {
        int GPU_ID, GPU_BlockNum, GPU_rBegin;
        hipSetDevice( omp_get_thread_num() );
        hipGetDevice(&GPU_ID);
        if(GPU_ID){
            GPU_rBegin = round / deviceNum;
            GPU_BlockNum = GPU_rBegin + round % deviceNum;
        }else{
            GPU_rBegin = 0;
            GPU_BlockNum = round/deviceNum;
        }
        int GPU_rEnd = GPU_rBegin + GPU_BlockNum;
        dim3 grid3(round, GPU_BlockNum);

        int offset = GPU_rBegin * blockSize * vertexPadded;
        hipMalloc((void**) &distance_dev[GPU_ID], sizeof(int) * vertexPadded * vertexPadded);
        hipMemcpy((void*) &(distance_dev[GPU_ID][offset]), (void*) &distance_host[offset], sizeof(int) * vertexPadded * blockSize * GPU_BlockNum, hipMemcpyHostToDevice);

        #pragma omp barrier

        for (int r = 0; r < round; ++r) {

            int index = r * blockSize * vertexPadded;
			if(r >= GPU_rBegin && r < GPU_rEnd){
                if( !GPU_ID ) hipMemcpy((void*) &(distance_dev[1][index]), (void*) &(distance_dev[0][index]), sizeof(int) * vertexPadded * blockSize, hipMemcpyDeviceToDevice);
                else          hipMemcpy((void*) &(distance_dev[0][index]), (void*) &(distance_dev[1][index]), sizeof(int) * vertexPadded * blockSize, hipMemcpyDeviceToDevice);
            }
            #pragma omp barrier

            FW1<<< 1, block, blockSize * blockSize * sizeof(int) >>>(distance_dev[GPU_ID], r, vertexPadded);
            FW2<<< grid2, block, 3 * blockSize * blockSize * sizeof(int) >>>(distance_dev[GPU_ID], r, vertexPadded, round);
            FW3<<< grid3, block, 3 * blockSize * blockSize * sizeof(int) >>>(distance_dev[GPU_ID], r, vertexPadded, round, GPU_rBegin, GPU_ID);
            hipDeviceSynchronize();
        }
        hipMemcpy((void*) &distance_host[offset], (void*) &(distance_dev[GPU_ID][offset]), sizeof(int) * vertexPadded * blockSize * GPU_BlockNum, hipMemcpyDeviceToHost);
    }
    
    
    //write answer to output file
    std::ofstream outputFile(argv[2], std::ios::out | std::ios::binary);
    for(int i=0; i<vertexNum; ++i){
        for(int j=0; j<vertexNum; ++j){
            outputFile.write( (char*)&distance_host[ i * vertexPadded + j ], 4);
        }
    }
    outputFile.close();
    
    hipFree(distance_host);
    hipFree(distance_dev);
    return 0;
}