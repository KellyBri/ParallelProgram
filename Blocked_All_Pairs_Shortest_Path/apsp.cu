
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <cmath>

#define INFINITE 1000000000
int* distance_host;
int* distance_dev;


__global__ void FW1(int *distance_dev, int r, int vertexPadded){

    int i = r * blockDim.x + threadIdx.y;
    int j = r * blockDim.x + threadIdx.x;
    int offset = i * vertexPadded;

    extern __shared__ int dist[];
    dist[threadIdx.y * blockDim.x + threadIdx.x] = distance_dev[ offset + j ];
    __syncthreads();

    for(int k=0; k<blockDim.x; ++k){
        dist[ threadIdx.y * blockDim.x + threadIdx.x ] = min(dist[ threadIdx.y * blockDim.x + threadIdx.x ], dist[ threadIdx.y * blockDim.x + k ] + dist[ k * blockDim.x + threadIdx.x ]);
        __syncthreads();
    }
    distance_dev[ offset + j ] = dist[threadIdx.y * blockDim.x + threadIdx.x];
}


__global__ void FW2(int *distance_dev, int r, int vertexPadded, int total_round){

    int block_i, block_j;
    if(blockIdx.y == 0){
        block_i = r;
        block_j = (blockIdx.x + r + 1) % total_round;
    }else{
        block_j = r;
        block_i = (blockIdx.x + r + 1) % total_round;
    }

    int i = block_i * blockDim.x + threadIdx.y;
    int j = block_j * blockDim.x + threadIdx.x;
    int offset = i * vertexPadded;
    int index = threadIdx.y * blockDim.x + threadIdx.x;
    int blockSize_squard = blockDim.x * blockDim.x;


    extern __shared__ int dist[];
    dist[index] = distance_dev[offset + j];
    dist[blockSize_squard + index] = distance_dev[offset + threadIdx.x + r * blockDim.x];
    dist[2*blockSize_squard + index] = distance_dev[(threadIdx.y + r * blockDim.x) * vertexPadded + j ];
    __syncthreads();

    
    for (int k = 0; k < blockDim.x; k++) {
        int ik = threadIdx.y * blockDim.x + blockSize_squard + k;
        int kj = k * blockDim.x + 2 * blockSize_squard + threadIdx.x;
        dist[index] = min(dist[index], dist[ik] + dist[kj]);
        __syncthreads();
    }
    distance_dev[offset + j] = dist[index];
}


__global__ void FW3(int *distance_dev, int r, int vertexPadded, int total_round){

    int block_i = (r + blockIdx.y + 1) % total_round;
    int block_j = (r + blockIdx.x + 1) % total_round;

    int i = block_i * blockDim.x + threadIdx.y;
    int j = block_j * blockDim.x + threadIdx.x;
    int offset = i * vertexPadded;

    int index = threadIdx.y * blockDim.x + threadIdx.x;
    int blockSize_squard = blockDim.x * blockDim.x;

    extern __shared__ int dist[];
    dist[index] = distance_dev[offset + j]; //block(i,j)
    dist[blockSize_squard + index] = distance_dev[offset + threadIdx.x + r * blockDim.x]; //block(i,r)
    dist[2*blockSize_squard + index] = distance_dev[(threadIdx.y + r * blockDim.x) * vertexPadded + j ]; //block(r,j)
    __syncthreads();

    
    for (int k = 0; k < blockDim.x; k++) {
        int ik = threadIdx.y * blockDim.x + blockSize_squard + k;
        int kj = k * blockDim.x + 2 * blockSize_squard + threadIdx.x;
        dist[index] = min(dist[index], dist[ik] + dist[kj]);
        __syncthreads();
    }
    distance_dev[offset + j] = dist[index];
}

void block_FW(int blockSize, int vertexNum, int vertexPadded) {
    int round = vertexPadded / blockSize;
    dim3 block(blockSize, blockSize);
    dim3 grid2(round-1, 2);
    dim3 grid3(round-1, round-1);
    hipMalloc(&distance_dev, sizeof(int) * vertexPadded * vertexPadded);
    hipMemcpy(distance_dev, distance_host, sizeof(int) * vertexPadded * vertexPadded, hipMemcpyHostToDevice);

	for (int r = 0; r < round; ++r) {
        FW1<<< 1, block, blockSize * blockSize * sizeof(int) >>>(distance_dev, r, vertexPadded);
        FW2<<< grid2, block, 3 * blockSize * blockSize * sizeof(int) >>>(distance_dev, r, vertexPadded, round);
        FW3<<< grid3, block, 3 * blockSize * blockSize * sizeof(int) >>>(distance_dev, r, vertexPadded, round);
    }
    hipMemcpy(distance_host, distance_dev, sizeof(int) * vertexPadded * vertexPadded, hipMemcpyDeviceToHost);
}


int main(int argc, char **argv){

    //get number of threads per block
    hipSetDevice(0);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int ThreadsPerBlock = (int) sqrt(prop.maxThreadsPerBlock);
    int blockSize = ThreadsPerBlock;
    
    //read input file
    std::ifstream inputFile(argv[1], std::ios::in | std::ios::binary);
    unsigned vertexNum, edgeNum;
    inputFile.read((char*)&vertexNum, 4);
    inputFile.read((char*)&edgeNum, 4);

    //calculate block number, vertex number in a block
    if(vertexNum < blockSize) blockSize = vertexNum;
    int blockNum = ceil( 1.0 * vertexNum / blockSize);
    int vertexPadded = blockSize * blockNum;


    //Allocate memory (pinned)
    hipHostMalloc(&distance_host, sizeof(int) * vertexPadded * vertexPadded, hipHostMallocDefault);

    for(unsigned i=0; i<vertexPadded; ++i){
        for(unsigned j=0; j<vertexPadded; ++j){
            if( i>=vertexNum || j>=vertexNum) distance_host[ i * vertexPadded + j ] = INFINITE;
            else if( i == j) distance_host[ i * vertexPadded + j ] = 0;
            else distance_host[ i * vertexPadded + j ] = INFINITE;
        }
    }

    int source, destination, weight;
    while( inputFile.read((char*)&source, 4) ){
        inputFile.read((char*)&destination, 4);
        inputFile.read((char*)&weight, 4);
        distance_host[ source * vertexPadded + destination ] = weight;
    }
    inputFile.close();


    block_FW(blockSize, vertexNum, vertexPadded);

    
    //write answer to output file
    std::ofstream outputFile(argv[2], std::ios::out | std::ios::binary);
    for(int i=0; i<vertexNum; ++i){
        for(int j=0; j<vertexNum; ++j){
            outputFile.write( (char*)&distance_host[ i * vertexPadded + j ], 4);
        }
    }
    outputFile.close();
    
    hipFree(distance_host);
    hipFree(distance_dev);
    return 0;
}